#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#include <GL/gl.h>
#include <GL/glut.h>

#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#define WIDTH 1920
#define HEIGHT 1080
#define PIXELS WIDTH * HEIGHT
#define BYTES_PER_PIXEL 4
#define PIXELS_SIZE PIXELS * BYTES_PER_PIXEL
#define PIXEL_DIV 1024
#define MAX_ITERS 200
#define MOVE_SPEED 10
#define KEY_ESC 27
#define UP 119
#define DOWN 115
#define LEFT 97
#define RIGHT 100

double *x, *y, *zoom;
uint8_t *image;

GLuint tex;
hipGraphicsResource_t cuda_resource;

__global__
void update_image(uint8_t *image, double *x, double *y, double *zoom) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < PIXELS) {
        int sx = i % WIDTH;
        int sy = i / WIDTH;
        double mx = *x + (sx - WIDTH / 2) / *zoom;
        double my = *y + (sy - HEIGHT / 2) / *zoom;
        double zx = mx;
        double zy = my;
        double zx2, zy2;
        float iters = 0;
        while (zx * zx + zy * zy <= 4.0) {
            zx2 = (zx * zx) - (zy * zy) + mx;
            zy2 = (2 * zx * zy) + my;
            zx = zx2;
            zy = zy2;
            iters++;
            if (iters >= MAX_ITERS) {
                break;
            }
        }
        float H = iters / MAX_ITERS * 210.0 + 15.0;
        float r, g, b;

        float s = 0.5;
        float v = 0.8;
        if (iters >= MAX_ITERS) v = 0.0;
        float C = s*v;
        float X = C*(1-abs(fmod(H/60.0, 2)-1));
        float m = v-C;
        if(H >= 0 && H < 60){
            r = C;
            g = X;
            b = 0;
        }
        else if(H >= 60 && H < 120){
            r = X;
            g = C;
            b = 0;
        }
        else if(H >= 120 && H < 180){
            r = 0;
            g = C;
            b = X;
        }
        else if(H >= 180 && H < 240){
            r = 0;
            g = X;
            b = C;
        }
        else if(H >= 240 && H < 300){
            r = X;
            g = 0;
            b = C;
        }
        else{
            r = C;
            g = 0;
            b = X;
        }
        r = (r+m)*255;
        g = (g+m)*255;
        b = (b+m)*255;

        image[4 * i] = r;
        image[4 * i + 1] = g;
        image[4 * i + 2] = b;
    }
}

__global__
void update_surface(hipSurfaceObject_t cuda_surface, uint8_t *image) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < PIXELS_SIZE) {
        int x = i % (BYTES_PER_PIXEL * WIDTH);
        int y = i / (BYTES_PER_PIXEL * WIDTH);
        surf2Dwrite<uint8_t>(image[i], cuda_surface, x, y);
    }
}

void invokeRenderingKernel(hipSurfaceObject_t cuda_surface) {
    update_image<<<PIXELS/PIXEL_DIV, PIXEL_DIV>>>(image, x, y, zoom);
    update_surface<<<PIXELS_SIZE/PIXEL_DIV, PIXEL_DIV>>>(cuda_surface, image);
}

void initializeGL () {
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, WIDTH, HEIGHT, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
    glBindTexture(GL_TEXTURE_2D, 0);
    hipGraphicsGLRegisterImage(&cuda_resource, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
}

void displayGL() {
    hipGraphicsMapResources(1, &cuda_resource);
    hipArray_t cuda_array;
    hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_resource, 0, 0);
    hipResourceDesc cuda_array_resource_desc;
    cuda_array_resource_desc.resType = hipResourceTypeArray;
    cuda_array_resource_desc.res.array.array = cuda_array;
    hipSurfaceObject_t cuda_surface;
    hipCreateSurfaceObject(&cuda_surface, &cuda_array_resource_desc);
    invokeRenderingKernel(cuda_surface);
    hipDestroySurfaceObject(cuda_surface);
    hipGraphicsUnmapResources(1, &cuda_resource);

    glBindTexture(GL_TEXTURE_2D, tex);
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(+1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(+1.0f, +1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, +1.0f);
    glEnd();
    glBindTexture(GL_TEXTURE_2D, 0);
    glutSwapBuffers();
}

__global__
void set_pos(int mx, int my, double *x, double *y, double *zoom) {
    double set_x = -((double) mx - WIDTH / 2) / *zoom;
    double set_y = ((double) my - HEIGHT / 2) / *zoom;
    *x -= set_x;
    *y -= set_y;
}

__global__
void move(float mx, float my, float mzoom, double *x, double *y, double *zoom) {
    *x += mx / *zoom * MOVE_SPEED;
    *y += my / *zoom * MOVE_SPEED;
    *zoom *= mzoom;
}

void mouse(int button, int state, int mx, int my) {
    if (button == 0 && state == GLUT_DOWN) {
        set_pos<<<1, 1>>>(mx, my, x, y, zoom);
    }
    else if ((button == 3) || (button == 4)) {
       if (state == GLUT_UP) return;
       float mzoom = (button == 3) ? 2.0 : 0.5;
       move<<<1, 1>>>(0.0, 0.0, mzoom, x, y, zoom);
    }
}

void keyboardGL (unsigned char key, int mousePositionX, int mousePositionY) {
    switch (key) {
        case KEY_ESC:
            exit(0);
            break;
        case UP:
            move<<<1, 1>>>(0.0, 1.0, 1.0, x, y, zoom);
            break;
        case DOWN:
            move<<<1, 1>>>(0.0, -1.0, 1.0, x, y, zoom);
            break;
        case LEFT:
            move<<<1, 1>>>(-1.0, 0.0, 1.0, x, y, zoom);
            break;
        case RIGHT:
            move<<<1, 1>>>(1.0, 0.0, 1.0, x, y, zoom);
            break;
        default:
            break;
    }
}

int main (int argc, char *argv[]) {

    srand(time(0));

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("Mandelbrot Set");
    glutDisplayFunc(displayGL);
    glutIdleFunc(displayGL);
    glutKeyboardFunc(keyboardGL);
    glutMouseFunc(mouse);
    initializeGL();

    int i;

    uint8_t *host_image = (uint8_t *) malloc(PIXELS_SIZE * sizeof(uint8_t));

    double host_x, host_y;
    double host_zoom = HEIGHT / 2;

    hipMalloc(&image, PIXELS_SIZE * sizeof(uint8_t));
    hipMalloc(&x, sizeof(double));
    hipMalloc(&y, sizeof(double));
    hipMalloc(&zoom, sizeof(double));

    for (i = 0; i < PIXELS_SIZE; i++) {
        host_image[i] = 0;
    }

    hipMemcpy(image, host_image, PIXELS_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(x, &host_x, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y, &host_y, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(zoom, &host_zoom, sizeof(double), hipMemcpyHostToDevice);

    glutMainLoop();

    return 0;
}
